#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipcub/hipcub.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <chrono>

#include <random>

#include "stdio.h"


constexpr int32_t WIDTH = 4;
constexpr int32_t DEPTH = 8;


__host__ __device__ int32_t reverse_idx(int32_t idx, int32_t width, int32_t depth) {
    auto out = 0;
    for (int32_t i=0; i<depth; ++i) {
        const auto tmp = idx;
        idx /= tmp;
        const auto rem = tmp - idx*width;
        out *= width;
        out += rem;
    }
    return out;
}


__host__ __device__ inline  int32_t compute_offset(const int32_t* levels, int32_t level) {
    int32_t result = 0;
    for (int32_t i=0; i<level; ++i) {
        result += levels[i];
    }
    return result;
}

template <typename I, typename J>
constexpr __host__ __device__ I round_up_div(I num, J div) {
    return (num + static_cast<I>(div) - 1) / static_cast<I>(div);
}

void ft_mul_host(float* __restrict__ pd_out,
              const float* __restrict__ pd_lhs,
              const float* __restrict__ pd_rhs,
              int32_t max_depth,
              const int32_t* levels) {

    for (int32_t out_deg = max_depth; out_deg >= 0; --out_deg) {
        auto* out_p = pd_out + compute_offset(levels, out_deg);

        for (int32_t lhs_deg = out_deg; lhs_deg >= 0; --lhs_deg) {
            const auto rhs_deg =  out_deg - lhs_deg;
            const auto* lhs_p = pd_lhs + compute_offset(levels, lhs_deg);
            const auto* rhs_p = pd_rhs + compute_offset(levels, rhs_deg);

            const auto lhs_n = levels[lhs_deg];
            const auto rhs_n = levels[rhs_deg];

            for (auto ix = 0; ix < lhs_n; ++ix) {
                for (auto jx = 0; jx < rhs_n; ++jx) {
                    out_p[ix*rhs_n + jx] += lhs_p[ix] * rhs_p[jx];
                }
            }
        }
    }
}

__global__ void
ft_mul_kernel(float* __restrict__ pd_out,
              const float* __restrict__ pd_lhs,
              const float* __restrict__ pd_rhs,
              int32_t max_depth,
              const int32_t* levels) {

    auto x_offset = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y_offset = blockIdx.y*blockDim.y + threadIdx.y;
    const auto step = gridDim.x * blockDim.x;
    const auto grid_step = gridDim.y*blockDim.y;


    for (int32_t out_deg = max_depth; out_deg >= 0; --out_deg) {
        auto* out_p = pd_out + compute_offset(levels, out_deg);

        auto y_set = grid_step;

        for (int32_t lhs_deg = out_deg; lhs_deg >= 0; --lhs_deg) {
            const auto rhs_deg =  out_deg - lhs_deg;
            const auto* lhs_p = pd_lhs + compute_offset(levels, lhs_deg);
            const auto* rhs_p = pd_rhs + compute_offset(levels, rhs_deg);

            __syncthreads();

            const auto lhs_n = levels[lhs_deg];
            const auto rhs_n = levels[rhs_deg];

//            __syncthreads();
            auto ix = x_offset;
            auto iy = y_offset;
            if (ix < lhs_n && iy < rhs_n) {
                out_p[ix * rhs_n + iy] += lhs_p[ix]*rhs_p[iy];
//                __syncthreads();
            }
//            for (auto ix = x_offset; ix < lhs_n; ix += step) {
//                auto *optr = out_p + ix * rhs_n;
//                const auto lhs_val = lhs_p[ix];
//                for (auto jx = y_offset; jx < rhs_n; jx += grid_step) {
//                    optr[jx] += lhs_val * rhs_p[jx];
//                }
//            }
//            __syncthreads();
        }
    }
}

template <typename T>
using rp_t = T* __restrict;
template <typename T>
using crp_t = const T* __restrict;

template <typename T>
struct WriteTensorData {
    rp_t<T> fwd_data;
    rp_t<T> rev_data;
};

template <typename T>
struct ReadTensorData {
    crp_t<T> fwd_read;
    crp_t<T> rev_read;
};

struct ComputeInfo {
    int32_t width;
    int32_t depth;
    int32_t tile_letters;
    const int32_t* levels;
    const int32_t* reverse_letters;
};


struct DivRem {
    int div;
    int rem;
};


__device__ __host__ DivRem divide(int idx, int divisor) {

    DivRem result;
    result.div = idx / divisor;
    result.rem = (idx - result.div * divisor);

    return result;
}


template <typename T>
__global__ void ft_tiled_mul(WriteTensorData<T> out,
                             ReadTensorData<T> lhs,
                             ReadTensorData<T> rhs,
                             ComputeInfo info) {
    const auto& xi = threadIdx.x;
    const auto& yi = threadIdx.y;
    const auto grid_x = gridDim.x * blockDim.x;
    const auto grid_y = gridDim.y * blockDim.y;
    const auto& tile_width = info.levels[info.tile_letters];

    const auto tile_idx = xi*blockDim.x + yi;

    auto get_offset = [&info] (int32_t level, int32_t offset) -> int32_t {
        auto level_offset = compute_offset(info.levels, level);
        return level_offset + offset;
    };


    extern __shared__ T tile[];   // size blockDim.x * blockDim.y
    const auto tile_size = tile_width * tile_width;

    T lhs_val = 0;
    T rhs_val = 0;

    for (int32_t out_deg=info.depth; out_deg >= 2*info.tile_letters; --out_deg) {
        const auto mid_deg = out_deg - 2 * info.tile_letters;
        const auto& mid_stride = info.levels[mid_deg];


        for (int32_t mid_idx=0; mid_idx < info.levels[mid_deg]; ++mid_idx) {
            const auto mid_ridx = reverse_idx(mid_idx, info.width, mid_deg);

            tile[tile_idx] = 0;
            __syncthreads();

            for (int32_t lhs_deg = 1; lhs_deg < info.tile_letters; ++lhs_deg) {
                auto rhs_deg = out_deg - lhs_deg;

                lhs_val = 0;
                rhs_val = 0;

                const auto& remainder_bound = info.levels[info.tile_letters + rhs_deg];

                auto split = divide(xi, remainder_bound);
                if (xi < tile_width && yi < tile_width) {
                    lhs_val = lhs.fwd_read[get_offset(lhs_deg, 0)];
                    rhs_val = rhs.fwd_read[get_offset(rhs_deg, (split.rem*mid_stride + mid_idx)*tile_width) + yi];
                }

                tile[tile_idx] += lhs_val*rhs_val;
            }

            for (int32_t lhs_mid_deg = 0; lhs_mid_deg <= mid_deg; ++lhs_mid_deg) {
                auto rhs_mid_deg = mid_deg - lhs_mid_deg;

                lhs_val = 0;
                rhs_val = 0;

                auto split = divide(mid_idx, info.levels[rhs_mid_deg]);
                if (xi < tile_width) {
                    lhs_val = lhs.fwd_read[get_offset(lhs_mid_deg + info.tile_letters,
                                           xi*info.levels[lhs_mid_deg] + split.div)];
                }
                if (yi < tile_width) {
                    rhs_val = rhs.fwd_read[get_offset(rhs_mid_deg + info.tile_letters,
                                           split.rem*tile_width + yi)];
                }

                tile[tile_idx] += lhs_val * rhs_val;
            }

            for (int32_t rhs_deg = 1; rhs_deg < info.tile_letters; ++rhs_deg) {
                auto lhs_deg = out_deg - rhs_deg;

                const auto small_bound = info.levels[lhs_deg];
                const auto &remainder_bound = info.levels[rhs_deg];
                lhs_val = 0;
                rhs_val = 0;

                auto split = divide(xi, remainder_bound);
                if (xi < tile_width && yi < tile_width) {
                    rhs_val = rhs.fwd_read[get_offset(rhs_deg,
                                                      (split.rem * mid_stride + mid_idx) * tile_width + yi)];
                    lhs_val = lhs.fwd_read[get_offset(lhs_deg, split.div)];
                }

                tile[tile_idx] += lhs_val * rhs_val;
            }

            if (xi < tile_width && yi < tile_width) {
                out.fwd_data[get_offset(out_deg, (xi*mid_stride + mid_idx)*tile_width + yi)]
                    += tile[tile_idx];
            }
        }





    }


}


int main() {

    thrust::host_vector<int32_t> powers;
    powers.reserve(1 + DEPTH);
    powers.push_back(1);
    thrust::host_vector<int32_t> offsets;
    offsets.reserve(1 + DEPTH);
    offsets.push_back(0);

    int32_t tensor_size = 1;
    for (int32_t i=1; i<=DEPTH; ++i) {
        offsets.push_back(offsets.back() + powers.back());
        powers.push_back(powers.back()*WIDTH);
        tensor_size = tensor_size * WIDTH + 1;
    }

    for (auto&& val : offsets) {
        std::cout << val << '\n';
    }

    thrust::device_vector<int32_t> device_powers(powers);
//    thrust::device_vector<int32_t> device_offsets(offsets);

    thrust::host_vector<float> in_left;
    thrust::host_vector<float> in_right;

    std::mt19937 rng(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    in_left.reserve(tensor_size);
    in_right.reserve(tensor_size);
    for (int32_t i=0; i<tensor_size; ++i) {
        in_left.push_back(dist(rng));
        in_right.push_back(dist(rng));
    }

    thrust::device_vector<float> din_left(in_left);
    thrust::device_vector<float> din_right(in_right);

    thrust::device_vector<float> dout(tensor_size);

    const int32_t* levels = thrust::raw_pointer_cast(&device_powers[0]);
    float* pd_out = thrust::raw_pointer_cast(&dout[0]);
    const float* pd_lhs = thrust::raw_pointer_cast(&din_left[0]);
    const float* pd_rhs = thrust::raw_pointer_cast(&din_right[0]);

    dim3 threads_per_block(32, 32);
    dim3 blocks { 1 };
    auto shared_size = threads_per_block.x * threads_per_block.y * sizeof(float);


    std::cout << "Blocks: " << blocks.x << ' ' << blocks.y << '\n';



    std::chrono::high_resolution_clock clk;
    auto start = clk.now();
    ft_mul_kernel<<<blocks, threads_per_block>>>(pd_out, pd_lhs, pd_rhs, DEPTH, levels);
//    ft_tiled_mul<float><<<blocks, threads_per_block, shared_size>>>(
        {pd_out, nullptr}, {pd_lhs, nullptr}, {pd_rhs, nullptr},
        {WIDTH, DEPTH, 2, levels, nullptr}
    );
    hipDeviceSynchronize();
    auto end = clk.now();

    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    std::cout << "Time: " << time.count() << '\n';



    thrust::host_vector<float> result(dout);

    thrust::host_vector<float> expected(tensor_size);
    ft_mul_host(expected.data(), in_left.data(), in_right.data(), DEPTH, powers.data());

    float err = 0.0f;
    float newerr;
    for (int32_t i=0; i<tensor_size; ++i) {
        if ((newerr = abs(expected[i]  - result[i])) > err) {
            std::cout << i << ' ' << expected[i] << ' ' << result[i] << ' ' << newerr << '\n';
            err = newerr;
        }
    }
    std::cout << "Max error: " << err << '\n';

    return 0;
}
