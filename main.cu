#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipcub/hipcub.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <chrono>

#include <random>

#include "stdio.h"


constexpr int32_t WIDTH = 4;
constexpr int32_t DEPTH = 8;


__host__ __device__ int32_t reverse_idx(int32_t idx, int32_t width, int32_t depth) {
    auto out = 0;
    for (int32_t i=0; i<depth; ++i) {
        const auto tmp = idx;
        idx /= tmp;
        const auto rem = tmp - idx*width;
        out *= width;
        out += rem;
    }
    return out;
}


__host__ __device__ inline  int32_t compute_offset(const uint32_t* levels, int32_t level) {
    int32_t result = 0;
    for (int32_t i=0; i<level; ++i) {
        result += levels[i];
    }
    return result;
}

template <typename I, typename J>
constexpr __host__ __device__ I round_up_div(I num, J div) {
    return (num + static_cast<I>(div) - 1) / static_cast<I>(div);
}

void ft_mul_host(float* __restrict__ pd_out,
              const float* __restrict__ pd_lhs,
              const float* __restrict__ pd_rhs,
              int32_t max_depth,
              const uint32_t* levels) {

    for (int32_t out_deg = max_depth; out_deg >= 0; --out_deg) {
        auto* out_p = pd_out + compute_offset(levels, out_deg);

        for (int32_t lhs_deg = out_deg; lhs_deg >= 0; --lhs_deg) {
            const auto rhs_deg =  out_deg - lhs_deg;
            const auto* lhs_p = pd_lhs + compute_offset(levels, lhs_deg);
            const auto* rhs_p = pd_rhs + compute_offset(levels, rhs_deg);

            const auto lhs_n = levels[lhs_deg];
            const auto rhs_n = levels[rhs_deg];

            for (auto ix = 0; ix < lhs_n; ++ix) {
                for (auto jx = 0; jx < rhs_n; ++jx) {
                    out_p[ix*rhs_n + jx] += lhs_p[ix] * rhs_p[jx];
                }
            }
        }
    }
}

__global__ void
ft_mul_kernel(float* __restrict__ pd_out,
              const float* __restrict__ pd_lhs,
              const float* __restrict__ pd_rhs,
              int32_t max_depth,
              const uint32_t* levels) {

    auto x_offset = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y_offset = blockIdx.y*blockDim.y + threadIdx.y;
    const auto step = gridDim.x * blockDim.x;
    const auto grid_step = gridDim.y*blockDim.y;


    for (int32_t out_deg = max_depth; out_deg >= 0; --out_deg) {
        auto* out_p = pd_out + compute_offset(levels, out_deg);

        auto y_set = grid_step;

        for (int32_t lhs_deg = out_deg; lhs_deg >= 0; --lhs_deg) {
            const auto rhs_deg =  out_deg - lhs_deg;
            const auto* lhs_p = pd_lhs + compute_offset(levels, lhs_deg);
            const auto* rhs_p = pd_rhs + compute_offset(levels, rhs_deg);

            __syncthreads();

            const auto lhs_n = levels[lhs_deg];
            const auto rhs_n = levels[rhs_deg];

            __syncthreads();
            auto ix = x_offset;
            auto iy = y_offset;
            if (ix < lhs_n && iy < rhs_n) {
                out_p[ix * rhs_n + iy] += lhs_p[ix]*rhs_p[iy];
            }
//            for (auto ix = x_offset; ix < lhs_n; ix += step) {
//                auto *optr = out_p + ix * rhs_n;
//                const auto lhs_val = lhs_p[ix];
//                for (auto jx = y_offset; jx < rhs_n; jx += grid_step) {
//                    optr[jx] += lhs_val * rhs_p[jx];
//                }
//            }
            __syncthreads();
        }
    }
}


int main() {

    thrust::host_vector<uint32_t> powers;
    powers.reserve(1 + DEPTH);
    powers.push_back(1);
    thrust::host_vector<int32_t> offsets;
    offsets.reserve(1 + DEPTH);
    offsets.push_back(0);

    int32_t tensor_size = 1;
    for (int32_t i=1; i<=DEPTH; ++i) {
        offsets.push_back(offsets.back() + powers.back());
        powers.push_back(powers.back()*WIDTH);
        tensor_size = tensor_size * WIDTH + 1;
    }

    for (auto&& val : offsets) {
        std::cout << val << '\n';
    }

    thrust::device_vector<uint32_t> device_powers(powers);
//    thrust::device_vector<int32_t> device_offsets(offsets);

    thrust::host_vector<float> in_left;
    thrust::host_vector<float> in_right;

    std::mt19937 rng(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    in_left.reserve(tensor_size);
    in_right.reserve(tensor_size);
    for (int32_t i=0; i<tensor_size; ++i) {
        in_left.push_back(dist(rng));
        in_right.push_back(dist(rng));
    }

    thrust::device_vector<float> din_left(in_left);
    thrust::device_vector<float> din_right(in_right);

    thrust::device_vector<float> dout(tensor_size);

    const uint32_t* levels = thrust::raw_pointer_cast(&device_powers[0]);
    float* pd_out = thrust::raw_pointer_cast(&dout[0]);
    const float* pd_lhs = thrust::raw_pointer_cast(&din_left[0]);
    const float* pd_rhs = thrust::raw_pointer_cast(&din_right[0]);

    dim3 threads_per_block(32, 32);
    dim3 blocks (round_up_div(powers.back(), threads_per_block.x), round_up_div(powers.back(), threads_per_block.y));

    std::cout << "Blocks: " << blocks.x << ' ' << blocks.y << '\n';

    std::chrono::high_resolution_clock clk;
    auto start = clk.now();
    ft_mul_kernel<<<blocks, threads_per_block>>>(pd_out, pd_lhs, pd_rhs, DEPTH, levels);
    hipDeviceSynchronize();
    auto end = clk.now();

    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    std::cout << "Time: " << time.count() << '\n';

    thrust::host_vector<float> result(dout);

    thrust::host_vector<float> expected(tensor_size);
    ft_mul_host(expected.data(), in_left.data(), in_right.data(), DEPTH, powers.data());

    float err = 0.0f;
    float newerr;
    for (int32_t i=0; i<tensor_size; ++i) {
        if ((newerr = abs(expected[i]  - result[i])) > err) {
            std::cout << i << ' ' << expected[i] << ' ' << result[i] << ' ' << newerr << '\n';
            err = newerr;
        }
    }
    std::cout << "Max error: " << err << '\n';

    return 0;
}
